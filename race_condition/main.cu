#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define N 32
#define NUM_ITERATIONS 1000

__global__ void kernel_with_race_condition(int *A) {
  A[0] = A[threadIdx.x]; // race condition
}

int main() {
  int *A = (int*)malloc(N*sizeof(int));
  for (int i = 0; i < N; i++) {
    A[i] = i;
  }

  int *d_A;
  hipMalloc(&d_A, N*sizeof(int));
  hipMemcpy(d_A, A, N*sizeof(int), hipMemcpyHostToDevice);

  for (int i = 0; i < NUM_ITERATIONS; i++) {
    kernel_with_race_condition<<<1, N>>>(d_A);
    int out;
    hipMemcpy(&out, d_A, sizeof(int), hipMemcpyDeviceToHost);
    assert(out >= 0 && out < N); // Select 1 of N values
  }
}